#include "hip/hip_runtime.h"
#include "activations.h"

// A more robust implementation for applying relu to a matrix
__global__ void relu_all(float *matrix, int m, int n) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int index = col * m + row;

    if (row < m && col < n) {
        if (matrix[index] < 0.0f) {
            matrix[index] = 0.0f;
        }
    }
}

//performs the relu operation on the whole mxn matrix
//assume each block is launched as 32 by 32 threads
//assume enough blocks are launched to tile the whole matrix
__global__ void relu_all_1D(float *matrix, int m, int n){
    int col = blockIdx.x % n / 32;
    int row = blockIdx.x * 32 / n;

    matrix[n * (row * 32 + threadIdx.y) + col * 32 + threadIdx.x] = (col * 32 + threadIdx.x < n && row * 32 + threadIdx.y < m) ? ((matrix[n * (row * 32 + threadIdx.y) + col * 32 + threadIdx.x] > 0.0) ? matrix[n * (row * 32 + threadIdx.y) + col * 32 + threadIdx.x] : 0.0) : 0.0;
}

//we say a vector is positive if its dot product of the vector of all ones is positive
//say it is a column major vector with 1024 elements per column
//launch 1024 x threads per block
//launch as many blocks in x as columns
__global__ void relu_vector(float *matrix_in float *matrix_out){
    int vector = 1024 * blockIdx.x;
    int t = threadIdx.x;
    __shared__ temp[1024] = {0};
    temp[t] = matrix_in[vector + t];
    __syncthreads();
    for(int i = 1; i < 1024; i*=2){
        if((t + 1) * 2 * i <= 1024){
            temp[(t+1) * 2 * i - 1] = temp[(t+1) * 2 * i - 1 - i] + temp[(t+1) * 2 * i - 1]; 
        }
    }
    __syncthreads();
    matrix_out[vector + t] = (temp[1023] >= 0.0) ? matrix_in[vector + t] : 0.0f;
}